
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void emptyKernel()
{
}

#define cudaCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      printf("CUDA error: %s - %s(%d)\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

int main(int argc, char* argv[])
{
	// Initialization
	/*----------------------------------------------------------------------------------------*/
	int device = atoi(argv[1]);
	cudaCheck(hipSetDevice(device));
	hipSetDeviceFlags(hipDeviceMapHost);

	int runtime_version;
	int driver_version;
	hipDeviceProp_t properties;
	hipGetDeviceProperties(&properties, device);
	hipRuntimeGetVersion(&runtime_version);
	hipDriverGetVersion(&driver_version);

	emptyKernel<<<1,1>>>();
	hipDeviceSynchronize();
	
	// Pool size
	/*----------------------------------------------------------------------------------------*/
	int pool_size = 1;
	char *h_data;
	break_01: __attribute__((unused));
	hipMallocManaged((void**)&h_data, 1 );
	 hipFree(h_data);
	
	// Maximum allocations and granularity
	/*----------------------------------------------------------------------------------------*/
	char **h_data_array = (char**) malloc(pool_size * sizeof(char*));
	hipMallocManaged((void**)&h_data_array[0], 1 );
	break_02: __attribute__((unused));
	int granularity = 0, iteration = 0, flag = 0;
	while(!flag && iteration < pool_size)
	{
		iteration++; 
		hipMallocManaged((void**)&h_data_array[iteration], 1 );
	}
	for(int i = 0; i <= iteration; i++)
	{
		 hipFree(h_data_array[i]);
	}
	free(h_data_array);
	
	// Size classes
	/*----------------------------------------------------------------------------------------*/
	char *h_data_inf, *h_data_sup;
	int inf_size = granularity, sup_size = granularity, finished = 1, class_finished = 0;
	break_03: __attribute__((unused));
	hipMallocManaged((void**)&h_data_inf, inf_size );
	while(!finished)
	{
		sup_size = sup_size + granularity;
		hipMallocManaged((void**)&h_data_sup, sup_size );
		 hipFree(h_data_sup);
		if(class_finished)
		{
			class_finished = 0;
			 hipFree(h_data_inf);
			inf_size = sup_size;
			hipMallocManaged((void**)&h_data_inf, inf_size );
		}
	}
	 hipFree(h_data_inf);
	
	// Larger allocations
	/*----------------------------------------------------------------------------------------*/
	break_04: __attribute__((unused));
	hipMallocManaged((void**)&h_data, pool_size + 1 );
	 hipFree(h_data);

	// Allocator policy
	/*----------------------------------------------------------------------------------------*/
	char *chunk_1, *chunk_2, *chunk_3, *chunk_4, *chunk_5, *chunk_6, *chunk_7, *chunk_8, *chunk_9, *chunk_10;
	hipMallocManaged((void**)&chunk_1, granularity * 2 );
	hipMallocManaged((void**)&chunk_2, granularity );
	hipMallocManaged((void**)&chunk_3, granularity * 2 );
	hipMallocManaged((void**)&chunk_4, granularity );
	hipMallocManaged((void**)&chunk_5, granularity );
	hipMallocManaged((void**)&chunk_6, granularity );
	 hipFree(chunk_1);
	 hipFree(chunk_3);
	 hipFree(chunk_5);
	hipMallocManaged((void**)&chunk_7, granularity );
	hipMallocManaged((void**)&chunk_8, granularity );
	break_05: __attribute__((unused));
	 hipFree(chunk_2);
	 hipFree(chunk_4);
	 hipFree(chunk_6);
	 hipFree(chunk_7);
	 hipFree(chunk_8);

	// Coalescing support
	/*----------------------------------------------------------------------------------------*/
	hipMallocManaged((void**)&chunk_1, granularity );
	hipMallocManaged((void**)&chunk_2, granularity );
	hipMallocManaged((void**)&chunk_3, granularity );
	 hipFree(chunk_1);
	 hipFree(chunk_2);
	hipMallocManaged((void**)&chunk_4, granularity * 2 );
	break_06: __attribute__((unused));
	 hipFree(chunk_3);
	 hipFree(chunk_4);

	// Splitting support
	/*----------------------------------------------------------------------------------------*/
	hipMallocManaged((void**)&chunk_1, granularity * 2 );
	hipMallocManaged((void**)&chunk_2, granularity );
	 hipFree(chunk_1);
	hipMallocManaged((void**)&chunk_3, granularity );
	break_07: __attribute__((unused));
	 hipFree(chunk_2);
	 hipFree(chunk_3);

	// Expansion policy
	/*----------------------------------------------------------------------------------------*/
	int max_allocations = pool_size / granularity;
	h_data_array = (char**) malloc(max_allocations * sizeof(char*));
	hipMallocManaged((void**)&h_data_array[0], granularity );
	break_08: __attribute__((unused));
	int index;
	for(index = 1; index < max_allocations; index++)
	{
		hipMallocManaged((void**)&h_data_array[index], granularity );
	}
	for(index = 0; index < max_allocations; index++)
	{
		 hipFree(h_data_array[index]);
	}
	free(h_data_array);


	// Pool usage
	/*----------------------------------------------------------------------------------------*/
	int quarter = pool_size / 4;
	hipMallocManaged((void**)&chunk_1, quarter );
	hipMallocManaged((void**)&chunk_2, quarter );
	hipMallocManaged((void**)&chunk_3, quarter );
	hipMallocManaged((void**)&chunk_4, quarter );
	hipMallocManaged((void**)&chunk_5, quarter );
	hipMallocManaged((void**)&chunk_6, quarter );
	hipMallocManaged((void**)&chunk_7, quarter );
	hipMallocManaged((void**)&chunk_8, quarter );
	hipMallocManaged((void**)&chunk_9, quarter );
	 hipFree(chunk_1);
	 hipFree(chunk_2);
	 hipFree(chunk_5);
	hipMallocManaged((void**)&chunk_10, quarter );
	break_09: __attribute__((unused));
	 hipFree(chunk_10);

	// Shrinking support
	/*----------------------------------------------------------------------------------------*/
	flag = 0;
	break_10: __attribute__((unused));
	 hipFree(chunk_6);
	 hipFree(chunk_7);
	 hipFree(chunk_8);
	flag = 1;
	 hipFree(chunk_9);
	flag = 2;
	 hipFree(chunk_3);
	 hipFree(chunk_4);

	// Finalization
	/*----------------------------------------------------------------------------------------*/
	hipDeviceReset();
	return 0;
}