
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

#define CHECK(call)															\
{																			\
	const hipError_t error = call;											\
	if(error != hipSuccess)												\
	{																		\
		printf("Error: %s:%d, ", __FILE__, __LINE__);						\
		printf("code:%d, reason: %s\n", error, hipGetErrorString(error));	\
		exit(1);															\
	}																		\
}		

__global__ void emptyKernel()
{
}

int main()
{
	//cudaSetDeviceFlags(cudaDeviceMapHost | cudaDeviceScheduleSpin);
	emptyKernel<<<1,1>>>();
	hipDeviceSynchronize();
	char *A = 0, *B = 0;
	int N;
	
	N = 1;
	std::cout << "First allocation of " << N << " B" << std::endl;
    hipMallocManaged(&A, N);
    std::cout << "Second allocation of " << N << " B" << std::endl;
	hipMallocManaged(&B, N);
	std::cout << "First free" << std::endl;
    hipFree(A);
    std::cout << "First free" << std::endl;
	hipFree(B);
	
	N = 65536;
    std::cout << "First allocation of " << N << " B" << std::endl;
    hipMallocManaged(&A, N);
    std::cout << "Second allocation of " << N << " B" << std::endl;
	hipMallocManaged(&B, N);
	std::cout << "First free" << std::endl;
    hipFree(A);
    std::cout << "First free" << std::endl;
	hipFree(B);

	N = 1048576;
	std::cout << "First allocation of " << N << " B" << std::endl;
    hipMallocManaged(&A, N);
    std::cout << "Second allocation of " << N << " B" << std::endl;
	hipMallocManaged(&B, N);
	std::cout << "First free" << std::endl;
    hipFree(A);
    std::cout << "First free" << std::endl;
	hipFree(B);

	N = 2097152;
	std::cout << "First allocation of " << N << " B" << std::endl;
    hipMallocManaged(&A, N);
    std::cout << "Second allocation of " << N << " B" << std::endl;
	hipMallocManaged(&B, N);
	std::cout << "First free" << std::endl;
    hipFree(A);
    std::cout << "First free" << std::endl;
	hipFree(B);

	N = 2097156;
	std::cout << "First allocation of " << N << " B" << std::endl;
    hipMallocManaged(&A, N);
    std::cout << "Second allocation of " << N << " B" << std::endl;
	hipMallocManaged(&B, N);
	std::cout << "First free" << std::endl;
    hipFree(A);
    std::cout << "First free" << std::endl;
	hipFree(B);


	std::cout << "----------------------------------------\n" ;

	hipDeviceReset();

}
